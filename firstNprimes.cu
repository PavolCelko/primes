// firstNprimes.cpp : Defines the entry point for the console application.
//
//#include "stdafx.h"
#include <math.h>
#include <stdio.h>
#include <string>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#define TWO				2
#define FIRST_PRIME		TWO

typedef struct
{

}primesClass_t;

__global__
void isPrime(const unsigned int number, const unsigned int *divisors, const unsigned int maxDivisor, unsigned long int *results)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// int stride = blockDim.x * gridDim.x;
	// int i;

	__syncthreads();

	results[index] = number % divisors[index];

	// for (i = index; divisors[i] <= maxDivisor; i += stride)
	// {
	// 	results[i] = number % divisors[i];
		
	// 	if (number % divisors[i])
	// 		continue;
	// 	else
	// 		break;
	// }

	__syncthreads();

	//return true;
	//results[i] = true;
}

void syncPrimes(const unsigned int *h_aPrimes, unsigned int *d_aPrimes, 
	const unsigned int *h_pNumOfPrimesFound, unsigned int *h_pNumOfPrimesFoundOnGPU)
{
	hipError_t err;
	
	// printf("h_pNumOfPrimesFound = %d\n", *h_pNumOfPrimesFound);
	// printf("h_pNumOfPrimesFoundOnGPU = %d\n", *h_pNumOfPrimesFoundOnGPU);

	err = hipMemcpy(&d_aPrimes[*h_pNumOfPrimesFoundOnGPU], &h_aPrimes[*h_pNumOfPrimesFoundOnGPU], 
		(*h_pNumOfPrimesFound - *h_pNumOfPrimesFoundOnGPU)*sizeof(unsigned int), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
        fprintf(stderr, "Sync primes failed (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
	}
	else
	{
		*h_pNumOfPrimesFoundOnGPU = *h_pNumOfPrimesFound;
		printf("GPU primes array synced\n");
	}
}

bool find_N_primes(const unsigned int N, unsigned int *h_aPrimes, unsigned long int uSize, unsigned int *h_pNumOfPrimesFound)
{
	unsigned int number;
	unsigned int maxDivisor;
	unsigned int *d_aPrimes = NULL;
	unsigned int *h_aPrimesTempHelp = NULL;
	unsigned int h_numOfPrimesFoundOnGPU = 0;

	hipError_t err;

	err = hipMalloc((void **)&d_aPrimes, uSize);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate GPU memory for primes array (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
	}
	h_aPrimesTempHelp = (unsigned int *)malloc(uSize);
	if(h_aPrimesTempHelp == NULL)
	{
		printf("Failed to allocate h_aPrimesTempHelp\n");
        exit(EXIT_FAILURE);
	}
	// printf("GPU primes array allocated\n");
	err = hipMemcpy(d_aPrimes, h_aPrimes, uSize, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
        fprintf(stderr, "Failed to copy array to GPU (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
	}
	printf("GPU primes array allocated\n");
		
	unsigned long int* d_results = NULL;
	err = hipMalloc((void **)&d_results, uSize);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate GPU memory for d_results (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
	}
	unsigned long int *h_pResults = (unsigned long int*)malloc(uSize);
	printf("GPU d_results allocated\n");

	int i, maxDivisorIndex;

	for (number = 3; *h_pNumOfPrimesFound < N; number++)
	{
		maxDivisor = (int)sqrt((double) number);
		for(maxDivisorIndex = 0; h_aPrimes[maxDivisorIndex] < maxDivisor; maxDivisorIndex++);
		// printf("h_pNumOfPrimesFound = %d\n", *h_pNumOfPrimesFound);
		if(number == 121)
		{
			printf("maxDivisor %d\n", maxDivisor);
			printf("maxDivisorIndex %d\n", maxDivisorIndex);
			printf("h_aPrimes[maxDivisorIndex] %u\n", h_aPrimes[maxDivisorIndex]);
			printf("h_numOfPrimesFoundOnGPU = %u\n", h_numOfPrimesFoundOnGPU);
			printf("*h_pNumOfPrimesFound = %u\n", *h_pNumOfPrimesFound);
			printf("h_aPrimes[h_numOfPrimesFoundOnGPU] %u\n", h_aPrimes[h_numOfPrimesFoundOnGPU]);
			hipDeviceSynchronize();
			err = hipMemcpy((void *)h_aPrimesTempHelp, (void *)d_aPrimes, 5*sizeof(unsigned int), hipMemcpyDeviceToHost);
			for (i = 0; i < h_numOfPrimesFoundOnGPU; i++)
				printf("sync d_aPrimes[%d] = %u\n", i, h_aPrimesTempHelp[i]);
		}
		if (maxDivisor > h_aPrimes[h_numOfPrimesFoundOnGPU])
			syncPrimes(h_aPrimes, d_aPrimes, h_pNumOfPrimesFound, &h_numOfPrimesFoundOnGPU);
		
		isPrime<<<1, 1024>>>(number, d_aPrimes, maxDivisor, d_results);
		hipDeviceSynchronize();
		err = hipMemcpy(h_pResults, d_results, uSize, hipMemcpyDeviceToHost);
		for(i = 0; i <= maxDivisorIndex; i++)
		{			
			
			if(number == 121)
			{	
				printf("h_aPrimesTempHelp[%d] = %u\n", i, h_aPrimesTempHelp[i]);
				printf("h_pResults[%d] = %ld\n", i, h_pResults[i]);				
			}
			if(h_pResults[i] == 0)
				break;
		}
		if (i == maxDivisorIndex + 1)
		{
			printf("PRIME FOUND = %d\n", number);
			h_aPrimes[(*h_pNumOfPrimesFound)++] = number;
		}					
	}

	// err = cudaMemcpy(h_aPrimes, d_aPrimes, uSize, cudaMemcpyDeviceToHost);
	// if (err != cudaSuccess)
	// {
    //     fprintf(stderr, "Failed to copy calculated primes from GPU to host (error code %s)!\n", cudaGetErrorString(err));
    //     exit(EXIT_FAILURE);
	// }
	
	hipFree(d_results);
	hipFree(d_aPrimes);
	free(h_aPrimesTempHelp);

	return true;
}


//int _tmain(int argc, _TCHAR* argv[])
int main(int argc, char* argv[])
{
	// printf("argc: %d\n", argc);
	// int i_dec = 0;
	// std::string::size_type sz;   // alias of size_t
	
	unsigned int uiFirst_N_primes = 10e2;
	
	if (argc > 1)
	{
		uiFirst_N_primes = atoi(argv[1]);		
		//char* str_dec = argv[1];
	}
	printf("find %d primes\n", uiFirst_N_primes);
	unsigned long int ulSize = uiFirst_N_primes * sizeof(unsigned int);
	printf("size = %lu\n", ulSize);
	unsigned int h_numOfPrimesFound = 0;
	unsigned int* h_aPrimes = (unsigned int*)malloc(ulSize);
	if (h_aPrimes == NULL)
	{
		printf("Not enough memory for %d primes.\n", uiFirst_N_primes);
		exit(EXIT_FAILURE);
	}
	memset(h_aPrimes, 1U, ulSize);		

	// evaluate 2 as prime manually
	h_aPrimes[h_numOfPrimesFound++] = FIRST_PRIME;
	
	find_N_primes(uiFirst_N_primes, h_aPrimes, ulSize, &h_numOfPrimesFound);
	
	FILE* fw = fopen("cudaListOfPrimes.txt", "w");
	unsigned int i;

	for (i = 0; i < uiFirst_N_primes; i++)
	{
		fprintf(fw, "%d\n", h_aPrimes[i]);
	}
	
	fclose(fw);
	free(h_aPrimes);

	return 0;
}

